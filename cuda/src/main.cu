/**
 * @file main.c
 * @brief Main file for the OpenMP implementation of the Game of Life
 * 
 * How to compile:
 * move to the openmp directory and run 'make' command.
 * Check the Makefile for more details.
 * 
 * How to run (from the openmp directory):
 * ./bin/multilayer-game-of-life <grid_size> <num_layers> <density> <num_steps> <seed>
 * The parameters are optional, if not provided, the default values are used.
 */
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <omp.h>


#include "vector_add.cuh"


#include "converter.h"
#include "ml_gol.h"

#define DEFAULT_GRID_SIZE 128
#define DEFAULT_NUM_LAYERS 3
#define DEFAULT_NUM_STEPS 64
#define DEFAULT_CREATE_PNG true
#define DEFAULT_DENSITY 0.3

int main(int argc, char *argv[]) {
    
    uint64_t grid_size = DEFAULT_GRID_SIZE;
    uint64_t num_layers = DEFAULT_NUM_LAYERS;
    uint64_t num_steps = DEFAULT_NUM_STEPS;
    bool create_png = DEFAULT_CREATE_PNG;
    float density = DEFAULT_DENSITY;
    uint64_t seed = time(NULL);

    if (argc > 1) {
        grid_size = atouint64(argv[1]);
    }

    if (argc > 2) {
        num_layers = atouint64(argv[2]);
    }

    if (argc > 3) {
        num_steps = atouint64(argv[3]);
    }

    if (argc > 4) {
        create_png = atoi(argv[4]) != 0;
    }

    if (argc > 5) {
        density = atof(argv[4]);
    }

    if (argc > 6) {
        seed = atouint64(argv[4]);
    }

    if (grid_size == 0 || num_layers == 0 || num_steps == 0) {
        fprintf(stderr, "Invalid input\n");
        return EXIT_FAILURE;
    }


    /*####################################################################
    ########################    CUDA TEST   ##############################
    ######################################################################*/

    float *h_a, *h_b, *h_c; // host copies of a, b, c
    float *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(float);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Allocate space for host copies of a, b, c and setup input values
    h_a = (float *)malloc(size);
    h_b = (float *)malloc(size);
    h_c = (float *)malloc(size);
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    // Copy inputs to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Launch vectorAdd() kernel on GPU with N blocks
    vectorAdd<<<(N + 255) / 256, 256>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

    // Print the result
    for (int i = 0; i < N; i++) {
        if (i % 100 == 0) {
            printf("h_c[%d] = %f\n", i, h_c[i]);
        }
    }

    // Cleanup
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    /* 
    printf("Starting Multilayer Game of Life.\n");
    printf("Max num of threads: %d\n",  omp_get_max_threads());

    double tstart, tstop;
    tstart = omp_get_wtime();

    start_game(grid_size, num_layers, num_steps, create_png, density, seed);

    tstop = omp_get_wtime();
    printf("Elapsed time: %f\n", tstop - tstart); 
    */

    return EXIT_SUCCESS;
}