#include "hip/hip_runtime.h"
extern "C" {
#include "hpc.h"
#include "image.h"
#include "color.h"

#include <stdlib.h>
#include <stdio.h>

}

#include "ml_gol.cuh"

#define BLKDIM 32

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


void start_game_on_cuda(const uint64_t grid_size, const uint64_t num_layers, const uint64_t num_steps, const bool create_png, const float density, const uint64_t seed) {
    ml_gol_t* ml_gol;
    ml_gol = (ml_gol_t*) malloc(sizeof(ml_gol_t));
    init_ml_gol(ml_gol, grid_size, num_layers, density, seed);
    /* hipMallocManaged(&ml_gol, sizeof(ml_gol_t));
    init_ml_gol_managed(ml_gol, grid_size, num_layers, density, seed); */
    
    bool* d_current = nullptr;
    bool* d_next = nullptr;
    color_t* d_layers_colors = nullptr;
    color_t* d_combined = nullptr;
    color_t* d_dependent = nullptr;
    init_ml_gol_to_device(ml_gol, &d_current, &d_next, &d_layers_colors, &d_combined, &d_dependent); 
    
    //TODO: BISOGNA CAPIRE SE VANNO CONSIDERATE LE GHOST CELLS NELLE DIMENSIONI DELLA GRIGLIA E DEL BLOCCO
    dim3 gridDim((grid_size + BLKDIM - 1) / BLKDIM, (grid_size + BLKDIM - 1) / BLKDIM);
    dim3 blockDim(BLKDIM, BLKDIM);

    for (uint64_t s = 1; s < num_steps; s++) {
        fill_ghost_cells(ml_gol);
        
        update_d_ml_gol(ml_gol, &d_current, &d_next, &d_combined, &d_dependent);

        ml_gol_kernel<<<gridDim, blockDim>>>(d_current, d_next, d_layers_colors, d_combined, d_dependent, grid_size, num_layers); cudaCheckError();
        hipDeviceSynchronize();
        copy_back_to_host(ml_gol, &d_current, &d_next, &d_combined, &d_dependent);
    
        swap_grids(ml_gol);

        if (create_png) {
            create_png_for_step(ml_gol, s);
        }

        reset_combined_and_dependent(ml_gol);

    }

    //hipFree(&ml_gol);
    //free_ml_gol(ml_gol, d_ml_gol);
}

void init_ml_gol_managed(ml_gol_t* ml_gol, const uint64_t grid_size, const uint64_t num_layers, const float density, const uint64_t seed) {
    srand(seed);

    ml_gol->num_layers = num_layers;
    ml_gol->grid_size = grid_size;

    uint64_t gol_grids_size = (grid_size + 2) * (grid_size + 2) * num_layers;
    hipMallocManaged(&ml_gol->current, gol_grids_size * sizeof(bool));
    hipMallocManaged(&ml_gol->next, gol_grids_size * sizeof(bool));


    // init non ghost cells
    for (uint64_t i = 1; i < grid_size + 1; i++) {
        for (uint64_t j = 1; j < grid_size + 1; j++) {
            for (uint16_t l = 0; l < ml_gol->num_layers; l++) {
                size_t cell_idx = idx_flat(ml_gol, i, j, l);
                ml_gol->current[cell_idx] = ((float) rand() / RAND_MAX) < density;
            }
        }
    }

    fill_ghost_cells(ml_gol);

    hipMallocManaged(&ml_gol->layers_colors, num_layers * sizeof(color_t));
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        ml_gol->layers_colors[i] = get_color_for_layer(i, num_layers);
    }

    size_t size = (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t);

    hipMallocManaged(&ml_gol->combined, size * sizeof(color_t));
    hipMallocManaged(&ml_gol->dependent, size * sizeof(color_t));

    printf("Initialized multilayer game of life with %ld layers and grid size %ld\n", num_layers, grid_size);
    
    print_layers_colors(ml_gol);
}

void init_ml_gol(ml_gol_t* ml_gol, const uint64_t grid_size, const uint64_t num_layers, const float density, const uint64_t seed) {
    srand(seed);

    ml_gol->num_layers = num_layers;
    ml_gol->grid_size = grid_size;

    uint64_t gol_grids_size = (grid_size + 2) * (grid_size + 2) * num_layers;
    ml_gol->current = (bool*) malloc(gol_grids_size * sizeof(bool));
    ml_gol->next = (bool*) malloc(gol_grids_size * sizeof(bool));

    // init non ghost cells
    for (uint64_t i = 1; i < grid_size + 1; i++) {
        for (uint64_t j = 1; j < grid_size + 1; j++) {
            for (uint16_t l = 0; l < ml_gol->num_layers; l++) {
                size_t cell_idx = idx_flat(ml_gol, i, j, l);
                ml_gol->current[cell_idx]= ((float) rand() / RAND_MAX) < density;
            }
        }
    }

    ml_gol->layers_colors = (color_t*) malloc(num_layers * sizeof(color_t));
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        ml_gol->layers_colors[i] = get_color_for_layer(i, num_layers);
    }

    size_t size = (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t);

    ml_gol->combined = (color_t*) malloc(size);
    ml_gol->dependent = (color_t*) malloc(size);

    printf("Initialized multilayer game of life with %ld layers and grid size %ld\n", num_layers, grid_size);
    
    print_layers_colors(ml_gol);
}

void init_ml_gol_to_device(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_layers_colors, color_t** d_combined, color_t** d_dependent) {
    // cuurent & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMalloc((void**)(d_current), gols_grid_size * sizeof(bool)));
    cudaSafeCall( hipMalloc((void**)(d_next), gols_grid_size * sizeof(bool)));

    // layers colors (we copy the colors here becuase they never change)
    cudaSafeCall( hipMalloc((void**)(d_layers_colors), ml_gol->num_layers * sizeof(color_t)));
    cudaSafeCall( hipMemcpy(*d_layers_colors, ml_gol->layers_colors, ml_gol->num_layers * sizeof(color_t), hipMemcpyHostToDevice));
    
    // combined and dependent
    cudaSafeCall( hipMalloc((void**)(d_combined), (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t)));
    cudaSafeCall( hipMalloc((void**)(d_dependent), (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t)));
}

void update_d_ml_gol(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_combined, color_t** d_dependent) {

    // current & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMemcpy(*d_current, ml_gol->current, gols_grid_size * sizeof(bool), hipMemcpyHostToDevice));
    cudaSafeCall( hipMemcpy(*d_next, ml_gol->next, gols_grid_size * sizeof(bool), hipMemcpyHostToDevice));

    // combined and dependent
    cudaSafeCall( hipMemcpy(*d_combined, ml_gol->combined, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyHostToDevice));
    cudaSafeCall( hipMemcpy(*d_dependent, ml_gol->dependent, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyHostToDevice));
}

void copy_back_to_host(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_combined, color_t** d_dependent) {
    // current & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMemcpy(ml_gol->current, *d_current, gols_grid_size * sizeof(bool), hipMemcpyDeviceToHost));
    cudaSafeCall( hipMemcpy(ml_gol->next, *d_next, gols_grid_size * sizeof(bool), hipMemcpyDeviceToHost));

    // combined and dependent
    cudaSafeCall( hipMemcpy(ml_gol->combined, *d_combined, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyDeviceToHost));
    cudaSafeCall( hipMemcpy(ml_gol->dependent, *d_dependent, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyDeviceToHost));
}

void create_png_for_step(const ml_gol_t* ml_gol, const uint64_t step) {
    create_png_for_grid(ml_gol->combined, ml_gol->grid_size, step, "combined");
    create_png_for_grid(ml_gol->dependent, ml_gol->grid_size, step, "dependent");
}

void create_png_for_grid(const color_t* grid, const uint64_t grid_size, const uint64_t step, const char* folder) {
    char filename[50];

    // 3 channels: RGB
    const uint8_t channels = 3;
    uint8_t* buffer = (uint8_t*) malloc(grid_size * grid_size * channels * sizeof(uint8_t));

    for (uint64_t i = 0; i < grid_size; i++) {
        for (uint64_t j = 0; j < grid_size; j++) {
            uint64_t idx = (i * grid_size + j);

            buffer[idx * channels] =     grid[idx].r;
            buffer[idx * channels + 1] = grid[idx].g;
            buffer[idx * channels + 2] = grid[idx].b;
        }
    }

    sprintf(filename, "output/%s/%s%04ld.png", folder, folder, step);
    write_png_file(filename, grid_size, grid_size, buffer);

    free(buffer);
}

void print_layers_colors(const ml_gol_t* ml_gol) {
    char hex[8];
    printf("Colors for the layers:\n");
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        color_to_hex(ml_gol->layers_colors[i], hex);
        printf("Layer %ld: %s\n", i, hex);
    }
}

void reset_combined_and_dependent(ml_gol_t* ml_gol) {
    for (uint64_t i = 0; i < ml_gol->grid_size * ml_gol->grid_size; i++) {
        ml_gol->combined[i] = BLACK;
        ml_gol->dependent[i] = BLACK;
    }
}

void fill_ghost_cells(ml_gol_t* ml_gol) {
    const uint64_t TOP = 1;
    const uint64_t BOTTOM = ml_gol->grid_size - 1;
    const uint64_t LEFT = 1;
    const uint64_t RIGHT = ml_gol->grid_size - 1;
    const uint64_t HALO_TOP = TOP - 1;
    const uint64_t HALO_BOTTOM = BOTTOM + 1;
    const uint64_t HALO_LEFT = LEFT - 1;
    const uint64_t HALO_RIGHT = RIGHT + 1;

    for (uint64_t l = 0; l < ml_gol->num_layers; l++) {
        // Left and right borders
        for (uint64_t i = TOP; i < BOTTOM + 1; i++) {
            ml_gol->current[idx_flat(ml_gol, i, HALO_LEFT, l)]  = ml_gol->current[idx_flat(ml_gol, i, RIGHT, l)];
            ml_gol->current[idx_flat(ml_gol, i, HALO_RIGHT, l)] = ml_gol->current[idx_flat(ml_gol, i, LEFT, l)];
        }

        // Top and bottom borders
        for (uint64_t j = LEFT; j < RIGHT + 1; j++) {
            ml_gol->current[idx_flat(ml_gol, HALO_TOP, j, l)]    = ml_gol->current[idx_flat(ml_gol, BOTTOM, j, l)];
            ml_gol->current[idx_flat(ml_gol, HALO_BOTTOM, j, l)] = ml_gol->current[idx_flat(ml_gol, TOP, j, l)];
        }
        
        // Corners
        ml_gol->current[idx_flat(ml_gol, HALO_TOP, HALO_LEFT, l)]     = ml_gol->current[idx_flat(ml_gol, BOTTOM, RIGHT, l)];
        ml_gol->current[idx_flat(ml_gol, HALO_TOP, HALO_RIGHT, l)]    = ml_gol->current[idx_flat(ml_gol, BOTTOM, LEFT, l)];
        ml_gol->current[idx_flat(ml_gol, HALO_BOTTOM, HALO_LEFT, l)]  = ml_gol->current[idx_flat(ml_gol, TOP, RIGHT, l)];
        ml_gol->current[idx_flat(ml_gol, HALO_BOTTOM, HALO_RIGHT, l)] = ml_gol->current[idx_flat(ml_gol, TOP, LEFT, l)];
    }

}

color_t get_color_for_layer(const uint64_t layer, const uint64_t num_layers) {
    // Calculate the angle for the hue based on the layer
    double hue = (double) layer / num_layers * 360.0;

    color_hsv_t hsv_color;
    hsv_color.h = hue;
    hsv_color.s = 1.0; 
    hsv_color.v = 1.0;

    return hsv_to_rgb(hsv_color);
}

void swap_grids(ml_gol_t* ml_gol) {
    bool* temp = ml_gol->current;
    ml_gol->current = ml_gol->next;
    ml_gol->next = temp;
}

__host__ __device__ size_t idx(const ml_gol_t* ml_gol, uint64_t i, uint64_t j) {
    return (i * (ml_gol->grid_size + 2)) + j;
}

__host__ __device__ size_t idx_flat(const ml_gol_t* ml_gol, uint64_t i, uint64_t j, uint64_t layer) {
    return (((i * (ml_gol->grid_size + 2)) + j) * ml_gol->num_layers) + layer;
}

__device__ uint8_t count_layer_alive_neighbors(ml_gol_t* ml_gol, uint64_t i, uint64_t j, uint64_t layer) {
    return  ml_gol->current[idx_flat(ml_gol, i - 1, j - 1, layer)] + ml_gol->current[idx_flat(ml_gol, i - 1, j, layer)] + ml_gol->current[idx_flat(ml_gol, i - 1, j + 1, layer)] +
            ml_gol->current[idx_flat(ml_gol, i, j - 1, layer)]     +                                                      ml_gol->current[idx_flat(ml_gol, i, j + 1    , layer)] +
            ml_gol->current[idx_flat(ml_gol, i + 1, j - 1, layer)] + ml_gol->current[idx_flat(ml_gol, i + 1, j, layer)] + ml_gol->current[idx_flat(ml_gol, i + 1, j + 1, layer)];
}

__device__ color_t add_colors_device(const color_t c1, const color_t c2) {
    color_t result;
    result.r = (c1.r + c2.r) > 255 ? 255 : c1.r + c2.r;
    result.g = (c1.g + c2.g) > 255 ? 255 : c1.g + c2.g;
    result.b = (c1.b + c2.b) > 255 ? 255 : c1.b + c2.b;
    return result;
}

__global__ void ml_gol_kernel(bool* d_current, bool* d_next, color_t* d_layers_colors, color_t* d_combined, color_t* d_dependent, uint64_t grid_size, uint64_t num_layers) {
    //extern __shared__ ml_cell_t blk_cells[];
    
    // actual index of grid with NO ghost cells
    uint64_t x = threadIdx.x + (blockDim.x * blockIdx.x);
    uint64_t y = threadIdx.y + (blockDim.y * blockIdx.y);

    ml_gol_t* ml_gol = (ml_gol_t*) malloc(sizeof(ml_gol_t)); 
    ml_gol->current = d_current;
    ml_gol->next = d_next;
    ml_gol->layers_colors = d_layers_colors;
    ml_gol->combined = d_combined;
    ml_gol->dependent = d_dependent;
    ml_gol->grid_size = grid_size;
    ml_gol->num_layers = num_layers;


    size_t no_ghost_idx = x * ml_gol->grid_size + y;

    // since we have ghost cells, we need to avoid them
    // this should do the trick
    x++; y++;

    uint16_t tot_alive_neighbors = 0;

    for (uint64_t l = 0; l < ml_gol->num_layers; l++) {
        uint8_t alive_neighbors = count_layer_alive_neighbors(ml_gol, x, y, l);
        tot_alive_neighbors += alive_neighbors;

        // The state of the current cell
        bool is_alive = ml_gol->current[idx_flat(ml_gol, x, y, l)];

        // The state of the current cell in the next step based on the rules of the game of life
        bool next_state = (is_alive && !(alive_neighbors < 2 || alive_neighbors > 3)) || (!is_alive && alive_neighbors == 3);

        ml_gol->next[idx_flat(ml_gol, x, y, l)] = next_state;
        
        // COMBINED
        if (is_alive) ml_gol->combined[no_ghost_idx] = add_colors_device(ml_gol->combined[no_ghost_idx], ml_gol->layers_colors[l]);
    }

    //DEPENDENT
    uint8_t channel_value = (uint8_t) ((((float) tot_alive_neighbors) / 9) * 255);
    ml_gol->dependent[no_ghost_idx] = (color_t){channel_value, channel_value, channel_value};
    
    free(ml_gol);
}
