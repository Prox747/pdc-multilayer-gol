#include "hip/hip_runtime.h"
extern "C" {
#include "hpc.h"
#include "image.h"
#include "color.h"

#include <stdlib.h>
#include <stdio.h>

}

#include "ml_gol.cuh"

#define BLKDIM 32

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


void start_game_on_cuda(const uint64_t grid_size, const uint64_t num_layers, const uint64_t num_steps, const bool create_png, const float density, const uint64_t seed) {
    ml_gol_t* ml_gol;
    ml_gol = (ml_gol_t*) malloc(sizeof(ml_gol_t));
    init_ml_gol(ml_gol, grid_size, num_layers, density, seed);

    /**
     * Uncomment to use managed unified memory.
     * Automatically manages memory transfers between GPU and CPU.
     * Simpler implementation but slower, especially with our data structures.
    */
    /* hipMallocManaged(&ml_gol, sizeof(ml_gol_t));
    init_ml_gol_managed(ml_gol, grid_size, num_layers, density, seed); */
    
    bool* d_current = nullptr;
    bool* d_next = nullptr;
    color_t* d_layers_colors = nullptr;
    color_t* d_combined = nullptr;
    color_t* d_dependent = nullptr;
    init_ml_gol_to_device(ml_gol, &d_current, &d_next, &d_layers_colors, &d_combined, &d_dependent); 
    
    // we create blocks of 1024 threads, one thread per cell (considering all layers)
    dim3 gridDim((grid_size + BLKDIM - 1) / BLKDIM, (grid_size + BLKDIM - 1) / BLKDIM);
    dim3 blockDim(BLKDIM, BLKDIM);

    if (create_png) {
        create_all_pngs(ml_gol, &d_current, &d_next, &d_layers_colors, &d_combined, &d_dependent, num_steps, gridDim, blockDim);
    } else {
        /* Insert this as third argument to use shared memory (remember changes in kernel)
        (BLKDIM + 2) * (BLKDIM + 2) * num_layers
        */
        fill_ghost_cells(ml_gol->current, ml_gol->grid_size, ml_gol->num_layers);
        update_d_ml_gol(ml_gol, &d_current, &d_next, &d_combined, &d_dependent);

        /* hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, 0);
        printf("Max threads per block: %d VS threads launched per block: %d\n", prop.maxThreadsPerBlock, blockDim.x * blockDim.y * blockDim.z);
        printf("Max blocks in each dimension: %d, %d, %d VS blocks launched: %d, %d. %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2], gridDim.x, gridDim.y, gridDim.z);
        printf("Max registers per block: %d VS register used IDK\n", prop.regsPerBlock);
        printf("Max shared memory per block: %zu VS sh memory used (if inserted): %zu\n", prop.sharedMemPerBlock, (BLKDIM + 2) * (BLKDIM + 2) * num_layers * sizeof(bool)); */

        for (uint64_t s = 1; s < num_steps; s++) {
            ml_gol_kernel_one_step<<<gridDim, blockDim>>>(d_current, d_next, d_layers_colors, d_combined, d_dependent, grid_size, num_layers); cudaCheckError();
            hipDeviceSynchronize();

            swap_grids_no_ghost_kernel<<<gridDim, blockDim>>>(d_current, d_next, grid_size, num_layers);
            hipDeviceSynchronize();

            manage_ghost_cells_kernel<<<1,1>>>(d_current, d_next, grid_size, num_layers);
            hipDeviceSynchronize();
        }
        
        //to print last step uncomment this
        /* ml_gol_kernel_one_step<<<gridDim, blockDim>>>(d_current, d_next, d_layers_colors, d_combined, d_dependent, grid_size, num_layers); cudaCheckError();
        hipDeviceSynchronize();

        copy_back_to_host(ml_gol, &d_current, &d_next, &d_combined, &d_dependent);

        swap_grids(ml_gol);

        create_png_for_step(ml_gol, num_steps); */
        
    }

    //hipFree(&ml_gol);
    free_ml_gol(ml_gol);
}

void free_ml_gol(ml_gol_t* ml_gol) {
    free(ml_gol->current);
    free(ml_gol->next);
    free(ml_gol->combined);
    free(ml_gol->dependent);
    free(ml_gol);
}

void init_ml_gol_managed(ml_gol_t* ml_gol, const uint64_t grid_size, const uint64_t num_layers, const float density, const uint64_t seed) {
    srand(seed);

    ml_gol->num_layers = num_layers;
    ml_gol->grid_size = grid_size;

    uint64_t gol_grids_size = (grid_size + 2) * (grid_size + 2) * num_layers;
    hipMallocManaged(&ml_gol->current, gol_grids_size * sizeof(bool));
    hipMallocManaged(&ml_gol->next, gol_grids_size * sizeof(bool));


    // init non ghost cells
    for (uint64_t i = 1; i < grid_size + 1; i++) {
        for (uint64_t j = 1; j < grid_size + 1; j++) {
            for (uint16_t l = 0; l < ml_gol->num_layers; l++) {
                size_t cell_idx = idx_flat(grid_size, num_layers, i, j, l);
                ml_gol->current[cell_idx] = ((float) rand() / RAND_MAX) < density;
            }
        }
    }

    fill_ghost_cells(ml_gol->current, grid_size, num_layers);

    hipMallocManaged(&ml_gol->layers_colors, num_layers * sizeof(color_t));
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        ml_gol->layers_colors[i] = get_color_for_layer(i, num_layers);
    }

    size_t size = (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t);

    hipMallocManaged(&ml_gol->combined, size * sizeof(color_t));
    hipMallocManaged(&ml_gol->dependent, size * sizeof(color_t));

    printf("Initialized multilayer game of life with %ld layers and grid size %ld\n", num_layers, grid_size);
    
    print_layers_colors(ml_gol);
}

void init_ml_gol(ml_gol_t* ml_gol, const uint64_t grid_size, const uint64_t num_layers, const float density, const uint64_t seed) {
    srand(seed);

    ml_gol->num_layers = num_layers;
    ml_gol->grid_size = grid_size;

    uint64_t gol_grids_size = (grid_size + 2) * (grid_size + 2) * num_layers;
    ml_gol->current = (bool*) malloc(gol_grids_size * sizeof(bool));
    ml_gol->next = (bool*) malloc(gol_grids_size * sizeof(bool));

    // init non ghost cells
    for (uint64_t i = 1; i < grid_size + 1; i++) {
        for (uint64_t j = 1; j < grid_size + 1; j++) {
            for (uint16_t l = 0; l < ml_gol->num_layers; l++) {
                size_t cell_idx = idx_flat(grid_size, num_layers, i, j, l);
                ml_gol->current[cell_idx]= ((float) rand() / RAND_MAX) < density;
            }
        }
    }

    ml_gol->layers_colors = (color_t*) malloc(num_layers * sizeof(color_t));
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        ml_gol->layers_colors[i] = get_color_for_layer(i, num_layers);
    }

    size_t size = (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t);

    ml_gol->combined = (color_t*) malloc(size);
    ml_gol->dependent = (color_t*) malloc(size);

    printf("Initialized multilayer game of life with %ld layers and grid size %ld\n", num_layers, grid_size);
    
    print_layers_colors(ml_gol);
}

void init_ml_gol_to_device(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_layers_colors, color_t** d_combined, color_t** d_dependent) {
    // cuurent & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMalloc((void**)(d_current), gols_grid_size * sizeof(bool)));
    cudaSafeCall( hipMalloc((void**)(d_next), gols_grid_size * sizeof(bool)));

    // layers colors (we copy the colors here becuase they never change)
    cudaSafeCall( hipMalloc((void**)(d_layers_colors), ml_gol->num_layers * sizeof(color_t)));
    cudaSafeCall( hipMemcpy(*d_layers_colors, ml_gol->layers_colors, ml_gol->num_layers * sizeof(color_t), hipMemcpyHostToDevice));
    
    // combined and dependent
    cudaSafeCall( hipMalloc((void**)(d_combined), (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t)));
    cudaSafeCall( hipMalloc((void**)(d_dependent), (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t)));
}

void update_d_ml_gol(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_combined, color_t** d_dependent) {

    // current & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMemcpy(*d_current, ml_gol->current, gols_grid_size * sizeof(bool), hipMemcpyHostToDevice));
    cudaSafeCall( hipMemcpy(*d_next, ml_gol->next, gols_grid_size * sizeof(bool), hipMemcpyHostToDevice));

    // combined and dependent
    cudaSafeCall( hipMemcpy(*d_combined, ml_gol->combined, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyHostToDevice));
    cudaSafeCall( hipMemcpy(*d_dependent, ml_gol->dependent, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyHostToDevice));
}

void copy_back_to_host(const ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_combined, color_t** d_dependent) {
    // current & next
    uint64_t gols_grid_size = (ml_gol->grid_size + 2) * (ml_gol->grid_size + 2) * ml_gol->num_layers;

    cudaSafeCall( hipMemcpy(ml_gol->current, *d_current, gols_grid_size * sizeof(bool), hipMemcpyDeviceToHost));
    cudaSafeCall( hipMemcpy(ml_gol->next, *d_next, gols_grid_size * sizeof(bool), hipMemcpyDeviceToHost));

    // combined and dependent
    cudaSafeCall( hipMemcpy(ml_gol->combined, *d_combined, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyDeviceToHost));
    cudaSafeCall( hipMemcpy(ml_gol->dependent, *d_dependent, (ml_gol->grid_size) * (ml_gol->grid_size) * sizeof(color_t), hipMemcpyDeviceToHost));
}

void create_all_pngs(ml_gol_t* ml_gol, bool** d_current, bool** d_next, color_t** d_layers_colors, color_t** d_combined, color_t** d_dependent, uint64_t num_steps, dim3 gridDim, dim3 blockDim) {
    /* hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    printf("Max threads per block: %d VS threads launched per block: %d\n", prop.maxThreadsPerBlock, blockDim.x * blockDim.y * blockDim.z);
    printf("Max blocks in each dimension: %d, %d, %d VS blocks launched: %d, %d. %d\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2], gridDim.x, gridDim.y, gridDim.z);
    printf("Max registers per block: %d VS register used IDK\n", prop.regsPerBlock);
    printf("Max shared memory per block: %zu\n", prop.sharedMemPerBlock); */

    for (uint64_t s = 1; s < num_steps; s++) {
        fill_ghost_cells(ml_gol->current, ml_gol->grid_size, ml_gol->num_layers);
        
        update_d_ml_gol(ml_gol, d_current, d_next, d_combined, d_dependent);

        ml_gol_kernel_one_step<<<gridDim, blockDim>>>(*d_current, *d_next, *d_layers_colors, *d_combined, *d_dependent, ml_gol->grid_size, ml_gol->num_layers); cudaCheckError();
        hipDeviceSynchronize();
        copy_back_to_host(ml_gol, d_current, d_next, d_combined, d_dependent);
    
        swap_grids(ml_gol);

        create_png_for_step(ml_gol, s);

        reset_combined_and_dependent(ml_gol);
    }
}

void create_png_for_step(const ml_gol_t* ml_gol, const uint64_t step) {
    create_png_for_grid(ml_gol->combined, ml_gol->grid_size, step, "combined");
    create_png_for_grid(ml_gol->dependent, ml_gol->grid_size, step, "dependent");
}

void create_png_for_grid(const color_t* grid, const uint64_t grid_size, const uint64_t step, const char* folder) {
    char filename[50];

    // 3 channels: RGB
    const uint8_t channels = 3;
    uint8_t* buffer = (uint8_t*) malloc(grid_size * grid_size * channels * sizeof(uint8_t));

    for (uint64_t i = 0; i < grid_size; i++) {
        for (uint64_t j = 0; j < grid_size; j++) {
            uint64_t idx = (i * grid_size + j);

            buffer[idx * channels] =     grid[idx].r;
            buffer[idx * channels + 1] = grid[idx].g;
            buffer[idx * channels + 2] = grid[idx].b;
        }
    }

    sprintf(filename, "output/%s/%s%04ld.png", folder, folder, step);
    write_png_file(filename, grid_size, grid_size, buffer);

    free(buffer);
}

void print_layers_colors(const ml_gol_t* ml_gol) {
    char hex[8];
    printf("Colors for the layers:\n");
    for (uint64_t i = 0; i < ml_gol->num_layers; i++) {
        color_to_hex(ml_gol->layers_colors[i], hex);
        printf("Layer %ld: %s\n", i, hex);
    }
}

void reset_combined_and_dependent(ml_gol_t* ml_gol) {
    for (uint64_t i = 0; i < ml_gol->grid_size * ml_gol->grid_size; i++) {
        ml_gol->combined[i] = BLACK;
        ml_gol->dependent[i] = BLACK;
    }
}

__host__ __device__ __forceinline__ void fill_ghost_cells(bool* current, uint64_t grid_size, uint64_t num_layers) {
    const uint64_t TOP = 1;
    const uint64_t BOTTOM = grid_size - 1;
    const uint64_t LEFT = 1;
    const uint64_t RIGHT = grid_size - 1;
    const uint64_t HALO_TOP = TOP - 1;
    const uint64_t HALO_BOTTOM = BOTTOM + 1;
    const uint64_t HALO_LEFT = LEFT - 1;
    const uint64_t HALO_RIGHT = RIGHT + 1;

    size_t src_idx;
    size_t dst_idx;
    
    // Left and right borders
    for (uint64_t i = TOP; i < BOTTOM + 1; i++) {
        src_idx = idx_flat(grid_size, num_layers, i, LEFT, 0);
        dst_idx = idx_flat(grid_size, num_layers, i, HALO_RIGHT, 0);
        memcpy(&current[dst_idx], &current[src_idx], num_layers * sizeof(bool));

        src_idx = idx_flat(grid_size, num_layers, i, RIGHT, 0);
        dst_idx = idx_flat(grid_size, num_layers, i, HALO_LEFT, 0);
        memcpy(&current[dst_idx], &current[src_idx], num_layers * sizeof(bool));
    }

    // Top and bottom borders
    for (uint64_t j = 0; j <= HALO_RIGHT; j++) {
        src_idx = idx_flat(grid_size, num_layers, TOP, j, 0);
        dst_idx = idx_flat(grid_size, num_layers, HALO_BOTTOM, j, 0);
        memcpy(&current[dst_idx], &current[src_idx], num_layers * sizeof(bool));
            
        src_idx = idx_flat(grid_size, num_layers, BOTTOM, j, 0);
        dst_idx = idx_flat(grid_size, num_layers, HALO_TOP, j, 0);
        memcpy(&current[dst_idx], &current[src_idx], num_layers * sizeof(bool));
    }
}

color_t get_color_for_layer(const uint64_t layer, const uint64_t num_layers) {
    // Calculate the angle for the hue based on the layer
    double hue = (double) layer / num_layers * 360.0;

    color_hsv_t hsv_color;
    hsv_color.h = hue;
    hsv_color.s = 1.0; 
    hsv_color.v = 1.0;

    return hsv_to_rgb(hsv_color);
}

void swap_grids(ml_gol_t* ml_gol) {
    bool* temp = ml_gol->current;
    ml_gol->current = ml_gol->next;
    ml_gol->next = temp;
}

__device__ __forceinline__ void swap_grid_cell(bool* current, bool* next, uint64_t x, uint64_t y, uint64_t num_layers, uint64_t grid_size) {
    bool temp;
    for (uint64_t l = 0; l < num_layers; l++) {
        size_t idx = idx_flat(grid_size, num_layers, x, y, l);
        temp = current[idx];
        current[idx] = next[idx];
        next[idx] = temp;
    }
}

__host__ __device__ size_t idx(const ml_gol_t* ml_gol, uint64_t i, uint64_t j) {
    return (i * (ml_gol->grid_size + 2)) + j;
}

__host__ __device__ __forceinline__ size_t idx_flat(const uint64_t grid_size, uint64_t num_layers, uint64_t i, uint64_t j, uint64_t layer) {
    return (((i * (grid_size + 2)) + j) * num_layers) + layer;
}

__device__ __forceinline__ uint8_t count_layer_alive_neighbors(bool* current, uint64_t grid_size, uint64_t num_layers, uint64_t i, uint64_t j, uint64_t layer) {
    return  current[idx_flat(grid_size, num_layers, i - 1, j - 1, layer)] + current[idx_flat(grid_size, num_layers, i - 1, j, layer)] + current[idx_flat(grid_size, num_layers, i - 1, j + 1, layer)] +
            current[idx_flat(grid_size, num_layers, i, j - 1,     layer)] +                                                              current[idx_flat(grid_size, num_layers, i, j + 1    , layer)] +
            current[idx_flat(grid_size, num_layers, i + 1, j - 1, layer)] + current[idx_flat(grid_size, num_layers, i + 1, j, layer)] + current[idx_flat(grid_size, num_layers, i + 1, j + 1, layer)];
}

__device__ __forceinline__ color_t add_colors_device(const color_t c1, const color_t c2) {
    color_t result;
    result.r = (c1.r + c2.r) > 255 ? 255 : c1.r + c2.r;
    result.g = (c1.g + c2.g) > 255 ? 255 : c1.g + c2.g;
    result.b = (c1.b + c2.b) > 255 ? 255 : c1.b + c2.b;
    return result;
}

__global__ void ml_gol_kernel_one_step(bool* d_current, bool* d_next, color_t* d_layers_colors, color_t* d_combined, color_t* d_dependent, uint64_t grid_size, uint64_t num_layers) {
    //extern __shared__ ml_cell_t blk_cells[];
    
    // actual index of grid with NO ghost cells
    uint64_t x = threadIdx.x + (blockDim.x * blockIdx.x);
    uint64_t y = threadIdx.y + (blockDim.y * blockIdx.y);

    size_t no_ghost_idx = x * grid_size + y;

    // since we have ghost cells, we need to avoid them
    // this should do the trick
    x++; y++;

    uint16_t tot_alive_neighbors = 0;

    // reset combined and dependent
    d_combined[no_ghost_idx] = BLACK;
    d_dependent[no_ghost_idx] = BLACK;

    for (uint64_t l = 0; l < num_layers; l++) {
        uint8_t alive_neighbors = count_layer_alive_neighbors(d_current, grid_size, num_layers, x, y, l);
        tot_alive_neighbors += alive_neighbors;

        // The state of the current cell
        bool is_alive = d_current[idx_flat(grid_size, num_layers, x, y, l)];
        tot_alive_neighbors += (uint16_t) is_alive;

        // The state of the current cell in the next step based on the rules of the game of life
        bool next_state = (is_alive && !(alive_neighbors < 2 || alive_neighbors > 3)) || (!is_alive && alive_neighbors == 3);

        d_next[idx_flat(grid_size, num_layers, x, y, l)] = next_state;
        
        // COMBINED
        if (next_state) d_combined[no_ghost_idx] = add_colors_device(d_combined[no_ghost_idx], d_layers_colors[l]);
    }

    //DEPENDENT
    uint8_t channel_value = (uint8_t) ((((float) tot_alive_neighbors) / 9) * 255);
    d_dependent[no_ghost_idx] = (color_t){channel_value, channel_value, channel_value};
}

__global__ void swap_grids_no_ghost_kernel(bool* d_current, bool* d_next, uint64_t grid_size, uint64_t num_layers) {
    // actual index of grid with NO ghost cells
    uint64_t x = threadIdx.x + (blockDim.x * blockIdx.x);
    uint64_t y = threadIdx.y + (blockDim.y * blockIdx.y);

    x++; y++;

    //swap grids (one cell per thread) - no ghost swap
    swap_grid_cell(d_current, d_next, x, y, num_layers, grid_size);        
}

__global__ void manage_ghost_cells_kernel(bool* d_current, bool* d_next, uint64_t grid_size, uint64_t num_layers) {
    // one thread swaps ghost cells and then update them
        for (uint64_t row = 0; row <= grid_size + 1; row++) {
            swap_grid_cell(d_current, d_next, row, 0, num_layers, grid_size);
            swap_grid_cell(d_current, d_next, row, grid_size + 1, num_layers, grid_size);   
        }

        for (uint64_t column = 1; column <= grid_size; column++) {
            swap_grid_cell(d_current, d_next, 0, column, num_layers, grid_size);
            swap_grid_cell(d_current, d_next, grid_size + 1, column, num_layers, grid_size);   
        }
        
        fill_ghost_cells(d_current, grid_size, num_layers);
}
